#include "hip/hip_runtime.h"
//
// Created by lucas on 11/04/19.
//

#include "../include/ReLU.cuh"

ReLU::ReLU() = default;

void ReLU::setInputProps(int num_dims, int const *dims, int size) {
    assert(num_dims > 0 && num_dims <= 4);
    // set input_dims
    input_dims[0] = dims[0];
    input_dims[1] = 1;
    for (int i = 1; i < num_dims; ++i) {
        input_dims[1] *= dims[i];
    }
    // set input_size
    input_size = size;

    // calculate output_dims
    copy(input_dims, input_dims+output_num_dims, output_dims);
    // calculate ouptut_size
    ouptut_size = input_size;
}

void ReLU::forward() {
    dim3 grid(this->input_dims[0] * ceil(input_dims[1]/128));
    forward_cuda<<<grid, 128>>>(this->d_in, this->d_out, this->input_dims[1]);
}

__global__ void forward_cuda(double* d_in, double* d_out, int input_dims_1) {
    
    // blockDim = 128
    // gridDim = batch_size * ceil(input_dims[1]/128) 

    int tx = threadIdx.x;   
    // int diff_tx = blockDim.x; // # 128
    int bx = blockIdx.x;    // no. of batch

    for(int i = tx; i<input_dims_1; i+=128) {
        d_out[bx * input_dims_1 + tx] = (d_in[bx * input_dims_1 + tx]>0) ? d_in[bx * input_dims_1 + tx]>0 : 0;
    }
}

Tensor<double> &ReLU::forward(Tensor<double> &input) {
    input_ = input;
    product_ = input.relu();

    return product_;
}

void ReLU::backprop() {
    backprop_cuda<<<this->input_dims[0], 32>>>(this->d_in, this->d_out, this->input_dims[1]);
}

__global__ void backprop_cuda(double* d_in, double* d_out, int input_dims_1) {

    // blockDim = 32
    // gridDim = batch_size

    int tx = threadIdx.x;   // no
    // int diff_tx = blockDim.x; // # 32
    int bx = blockIdx.x;    // no. of batch

    for(int i = tx; i<input_dims_1; i+=32) {
        d_in[bx * input_dims_1 + tx] = (d_out[bx * input_dims_1 + tx]>0) ? d_out[bx * input_dims_1 + tx]>0 : 0;
    }
}

Tensor<double> ReLU::backprop(Tensor<double> chainGradient, double learning_rate) {
    return chainGradient * input_.reluPrime();
}

void ReLU::load(FILE *file_model) {

}

void ReLU::save(FILE *file_model) {

}
